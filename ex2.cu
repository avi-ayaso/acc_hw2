#include "hip/hip_runtime.h"
#include <iostream>
#include <cuda/atomic>
#include "ex2.h"
#define LOG_N_SLOTS 4
#define N_STREAMS 64
#define N_REGS 32
#define STREAM_AVAILABLE -1
#define END_RUN -2
#define SHMEM_PER_BLOCK 2577
#define REGS_PER_BLOCK 28

__device__ void prefix_sum(int arr[], int arr_size) {
    int tid = threadIdx.x;
    int increment;

    for (int stride = 1; stride < min(blockDim.x, arr_size); stride *= 2) {
        if (tid >= stride && tid < arr_size) {
            increment = arr[tid - stride];
        }
        __syncthreads();
        if (tid >= stride && tid < arr_size) {
            arr[tid] += increment;
        }
        __syncthreads();
    }
}

// Example single-threadblock kernel for processing a single image.
// Feel free to change it.
__global__ void process_image_kernel(uchar *in, uchar *out) {
    __shared__ int histogram[256];
    __shared__ uchar map[256];

    int tid = threadIdx.x;

    if (tid < 256) {
        histogram[tid] = 0;
    }
    __syncthreads();

    for (int i = tid; i < IMG_HEIGHT * IMG_HEIGHT; i += blockDim.x)
        atomicAdd(&histogram[in[i]], 1);

    __syncthreads();

    prefix_sum(histogram, 256);

    if (tid < 256) {
        float map_value = float(histogram[tid]) / (IMG_WIDTH * IMG_HEIGHT);
        map[tid] = ((uchar)(N_COLORS * map_value)) * (256 / N_COLORS);
    }

    __syncthreads();

    for (int i = tid; i < IMG_WIDTH * IMG_HEIGHT; i += blockDim.x) {
        out[i] = map[in[i]];
    }
}




class streams_server : public image_processing_server
{
private:
    // TODO define stream server context (memory buffers, streams, etc...)
	hipStream_t streams[N_STREAMS];

    // Feel free to change the existing memory buffer definitions.
    int img_id_in_streams[N_STREAMS];
	uchar *dimg_in;
    uchar *dimg_out;

public:
    streams_server()
    {
        // TODO initialize context (memory buffers, streams, etc...)
		
		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			img_id_in_streams[stream_idx] = STREAM_AVAILABLE;
			CUDA_CHECK(hipStreamCreate(&streams[stream_idx]));
		}
		
        CUDA_CHECK( hipMalloc(&dimg_in, N_STREAMS * IMG_WIDTH * IMG_HEIGHT) );
        CUDA_CHECK( hipMalloc(&dimg_out, N_STREAMS * IMG_WIDTH * IMG_HEIGHT) );
    }

    ~streams_server() override
    {
        // TODO free resources allocated in constructor
		
		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			CUDA_CHECK(hipStreamDestroy(streams[stream_idx]));
		}
		
		CUDA_CHECK( hipFree(dimg_in) );
        CUDA_CHECK( hipFree(dimg_out) );
    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        // TODO place memory transfers and kernel invocation in streams if possible.
		
		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			if (img_id_in_streams[stream_idx] != STREAM_AVAILABLE) {
				continue;
			}
			img_id_in_streams[stream_idx] = img_id;
			int offset = stream_idx * IMG_WIDTH * IMG_HEIGHT;
			CUDA_CHECK( hipMemcpyAsync(dimg_in + offset, img_in, IMG_WIDTH * IMG_HEIGHT, hipMemcpyHostToDevice, streams[stream_idx]));
			process_image_kernel<<<1, 1024, 0, streams[stream_idx]>>>(dimg_in + offset, dimg_out + offset);
			CUDA_CHECK( hipMemcpyAsync(img_out, dimg_out + offset, IMG_WIDTH * IMG_HEIGHT, hipMemcpyDeviceToHost, streams[stream_idx]));
//			printf("img_id: %d was enqueued to stream idx: %d\n",img_id,stream_idx);
			return true;
		}
		return false;
    }

    bool dequeue(int *img_id) override
    {
        // TODO query (don't block) streams for any completed requests.

		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			hipError_t status = hipStreamQuery(streams[stream_idx]);
			switch (status) {
			case hipSuccess:
				*img_id = img_id_in_streams[stream_idx]; // TODO return the img_id of the request that was completed.
				if(*img_id == STREAM_AVAILABLE) {
					continue;
				}
				img_id_in_streams[stream_idx] = STREAM_AVAILABLE;
//				printf("img_id: %d finished and was dequeued from stream idx: %d\n",*img_id,stream_idx);
				return true;
			case hipErrorNotReady:
				continue;
			default:
				CUDA_CHECK(status);
			}
		}
		return false;
    }
};

std::unique_ptr<image_processing_server> create_streams_server()
{
    return std::make_unique<streams_server>();
}

typedef struct request{
    int img_id ;
    uchar* img_in;
    uchar* img_out;
} request_context;



template <size_t size>
class ring_buffer{
private:
    static const size_t N = 1 << size;
    request_context _mailbox[N];
    cuda::atomic<size_t> _head;
    cuda::atomic<size_t> _tail;
    request_context failure;
public:
    ring_buffer(){
        _head = 0;
        _tail = 0;
        failure.img_id = -1;
        for(size_t i = 0 ; i < N ; i++){
            _mailbox[i].img_id = -1;
        }
    }
    __device__ __host__ bool push(request_context data){
        size_t tail = _tail.load(cuda::memory_order_relaxed);
        if(tail - _head.load(cuda::memory_order_acquire) == N) return false;
        _mailbox[_tail % N] = data;
        _tail.store(tail + 1 , cuda::memory_order_release);
        return true;
    }
    __device__ __host__ request_context pop(){
        request_context item;
        size_t head = _head.load(cuda::memory_order_relaxed);
        if(_tail.load(cuda::memory_order_acquire) == head) return failure;
        item = _mailbox[_head % N];
        _head.store(head + 1 , cuda::memory_order_release);
        return item;
    }
};


__device__ void process_image(uchar *in, uchar *out) {
    __shared__ int histogram[256];
    __shared__ uchar map[256];

    int tid = threadIdx.x;

    if (tid < 256) {
        histogram[tid] = 0;
    }
    __syncthreads();

    for (int i = tid; i < IMG_HEIGHT * IMG_HEIGHT; i += blockDim.x)
        atomicAdd(&histogram[in[i]], 1);

    __syncthreads();

    prefix_sum(histogram, 256);

    if (tid < 256) {
        float map_value = float(histogram[tid]) / (IMG_WIDTH * IMG_HEIGHT);
        map[tid] = ((uchar)(N_COLORS * map_value)) * (256 / N_COLORS);
    }

    __syncthreads();

    for (int i = tid; i < IMG_WIDTH * IMG_HEIGHT; i += blockDim.x) {
        out[i] = map[in[i]];
    }
    __syncthreads();
}


__global__ void producer_consumer_kernel(ring_buffer<LOG_N_SLOTS>* cpu_to_gpu, ring_buffer<LOG_N_SLOTS>* gpu_to_cpu) {
    request_context req;
    __shared__ uchar* img_in;
    __shared__ uchar* img_out;
    __shared__ bool con;
    do{
        if(threadIdx.x == 0){
            req = cpu_to_gpu[blockIdx.x].pop();
            con = false;
            img_in = req.img_in;
            img_out = req.img_out;
            if(req.img_id == -1) con = true;
            else if(req.img_id == END_RUN) {
                return;
            }                
        }
        __syncthreads();
        if(con) continue;   
        process_image(img_in, img_out);
        if(threadIdx.x == 0){
            while(!gpu_to_cpu[blockIdx.x].push(req));
        }
        __syncthreads();
    }while(1);
}



int getNumOfBlocks(int threads) {
    hipDeviceProp_t prop;
    int min_limit;
    int n_used_block_regs = threads * REGS_PER_BLOCK;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    int shmem_block_limit = prop.sharedMemPerMultiprocessor / SHMEM_PER_BLOCK;
    int regs_block_limit = prop.regsPerMultiprocessor / n_used_block_regs;
    int thread_block_limit = prop.maxThreadsPerMultiProcessor / threads;
    min_limit = shmem_block_limit < regs_block_limit ? shmem_block_limit : regs_block_limit;
    min_limit = min_limit < thread_block_limit ? min_limit :thread_block_limit;
    return min_limit * prop.multiProcessorCount;
}

class queue_server : public image_processing_server
{
private:
    ring_buffer<LOG_N_SLOTS> *cpu_to_gpu;
    ring_buffer<LOG_N_SLOTS> *gpu_to_cpu;
    char* pinned_host_buffer;
    int n_thread_blocks;
public:
    queue_server(int threads)
    {
        
        // TODO initialize host state
        // TODO launch GPU producer-consumer kernel with given number of threads
        n_thread_blocks = getNumOfBlocks(threads);
        // Allocate pinned host buffer for two shared_memory instances
        CUDA_CHECK(hipHostMalloc(&pinned_host_buffer, 2 * n_thread_blocks * sizeof(ring_buffer<LOG_N_SLOTS>)));
        // Use placement new operator to construct our class on the pinned buffer
        cpu_to_gpu = new (pinned_host_buffer) ring_buffer<LOG_N_SLOTS>[n_thread_blocks];
        gpu_to_cpu = new (pinned_host_buffer + n_thread_blocks * sizeof(ring_buffer<LOG_N_SLOTS>)) ring_buffer<LOG_N_SLOTS>[n_thread_blocks]; 
        producer_consumer_kernel<<<n_thread_blocks , threads>>>(cpu_to_gpu, gpu_to_cpu);
        std::cout << "num_of_blocks" << n_thread_blocks << std::endl;
    }

    ~queue_server() override
    {
        request_context end_context;
        end_context.img_id = END_RUN;
        end_context.img_in = NULL;
        end_context.img_out = NULL;
        while(!cpu_to_gpu[0].push(end_context));
        CUDA_CHECK( hipDeviceSynchronize() );
        CUDA_CHECK( hipHostFree(pinned_host_buffer) );

    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        // TODO push new task into queue if possible
        request_context req;
        req.img_id = img_id;
        req.img_in = img_in;
        req.img_out = img_out;
        for(int i = 0 ; i < n_thread_blocks ; i++){
            if(cpu_to_gpu[i].push(req)){
                return true;
            } 
        }
        return false;
    }

    bool dequeue(int *img_id) override
    {
        // TODO query (don't block) the producer-consumer queue for any responses.
        *img_id = 0; // TODO return the img_id of the request that was completed.
        for(int i = 0 ; i < n_thread_blocks ; i++){         
            request_context req = gpu_to_cpu[i].pop();
            *img_id = req.img_id;
            if(req.img_id >= 0) {
                return true; 
            }
        }
        return false;
        
    }
};

std::unique_ptr<image_processing_server> create_queues_server(int threads)
{
    return std::make_unique<queue_server>(threads);
}
