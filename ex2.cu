#include "hip/hip_runtime.h"
#include <iostream>
#include "ex2.h"
#define N_SLOTS 16
#define N_STREAMS 64
#define STREAM_AVAILABLE -1
#define END_RUN -2

__device__ void prefix_sum(int arr[], int arr_size) {
    int tid = threadIdx.x;
    int increment;

    for (int stride = 1; stride < min(blockDim.x, arr_size); stride *= 2) {
        if (tid >= stride && tid < arr_size) {
            increment = arr[tid - stride];
        }
        __syncthreads();
        if (tid >= stride && tid < arr_size) {
            arr[tid] += increment;
        }
        __syncthreads();
    }
}

// Example single-threadblock kernel for processing a single image.
// Feel free to change it.
__global__ void process_image_kernel(uchar *in, uchar *out) {
    __shared__ int histogram[256];
    __shared__ uchar map[256];

    int tid = threadIdx.x;

    if (tid < 256) {
        histogram[tid] = 0;
    }
    __syncthreads();

    for (int i = tid; i < IMG_HEIGHT * IMG_HEIGHT; i += blockDim.x)
        atomicAdd(&histogram[in[i]], 1);

    __syncthreads();

    prefix_sum(histogram, 256);

    if (tid < 256) {
        float map_value = float(histogram[tid]) / (IMG_WIDTH * IMG_HEIGHT);
        map[tid] = ((uchar)(N_COLORS * map_value)) * (256 / N_COLORS);
    }

    __syncthreads();

    for (int i = tid; i < IMG_WIDTH * IMG_HEIGHT; i += blockDim.x) {
        out[i] = map[in[i]];
    }
}




class streams_server : public image_processing_server
{
private:
    // TODO define stream server context (memory buffers, streams, etc...)
	hipStream_t streams[N_STREAMS];

    // Feel free to change the existing memory buffer definitions.
    int img_id_in_streams[N_STREAMS];
	uchar *dimg_in;
    uchar *dimg_out;

public:
    streams_server()
    {
        // TODO initialize context (memory buffers, streams, etc...)
		
		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			img_id_in_streams[stream_idx] = STREAM_AVAILABLE;
			CUDA_CHECK(hipStreamCreate(&streams[stream_idx]));
		}
		
        CUDA_CHECK( hipMalloc(&dimg_in, IMG_WIDTH * IMG_HEIGHT) );
        CUDA_CHECK( hipMalloc(&dimg_out, IMG_WIDTH * IMG_HEIGHT) );
    }

    ~streams_server() override
    {
        // TODO free resources allocated in constructor
		
		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			CUDA_CHECK(hipStreamDestroy(streams[stream_idx]));
		}
		
		CUDA_CHECK( hipFree(dimg_in) );
        CUDA_CHECK( hipFree(dimg_out) );
    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        // TODO place memory transfers and kernel invocation in streams if possible.
		
		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			if (img_id_in_streams[stream_idx] != STREAM_AVAILABLE) {
				continue;
			}
			img_id_in_streams[stream_idx] = img_id;
			CUDA_CHECK( hipMemcpyAsync(dimg_in, img_in, IMG_WIDTH * IMG_HEIGHT, hipMemcpyHostToDevice, streams[stream_idx]));
			process_image_kernel<<<1, 1024, 0, streams[stream_idx]>>>(dimg_in, dimg_out);
			CUDA_CHECK( hipMemcpyAsync(img_out, dimg_out, IMG_WIDTH * IMG_HEIGHT, hipMemcpyDeviceToHost, streams[stream_idx]));
//			printf("img_id: %d was enqueued to stream idx: %d\n",img_id,stream_idx);
			return true;
		}
		return false;
    }

    bool dequeue(int *img_id) override
    {
        // TODO query (don't block) streams for any completed requests.

		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			hipError_t status = hipStreamQuery(streams[stream_idx]);
			switch (status) {
			case hipSuccess:
				*img_id = img_id_in_streams[stream_idx]; // TODO return the img_id of the request that was completed.
				if(*img_id == STREAM_AVAILABLE) {
					continue;
				}
				img_id_in_streams[stream_idx] = STREAM_AVAILABLE;
//				printf("img_id: %d finished and was dequeued from stream idx: %d\n",*img_id,stream_idx);
				return true;
			case hipErrorNotReady:
				continue;
			default:
				CUDA_CHECK(status);
			}
		}
		return false;
    }
};

std::unique_ptr<image_processing_server> create_streams_server()
{
    return std::make_unique<streams_server>();
}

typedef struct request{
    int img_id ;
    uchar* img_in;
    uchar* img_out;
} request_context;

typedef enum result {success , failure} result;

template <uint8_t size> class ring_buffer {
private:
    static const size_t N = 1 << size;
    request_context _mailbox[N] = {-1};
    cuda::atomic<size_t> _head = 0, _tail = 0;
public:
 result push(const request_context &data){
    int tail = _tail.load(memory_order_relaxed);
    if (tail - _head.load(memory_order_acquire) == N) return failure; // if queue is full
    _mailbox[_tail % N] = data;
    _tail.store(tail + 1, memory_order_release);
    return success;
 }
 request_context pop(){
    request_context item;
    item.img_id = -1; 
    int head = _head.load(memory_order_relaxed);
    if (_tail.load(memory_order_acquire) == _head) return item; // if queue is empty
    item = _mailbox[_head % N];
    _head.store(head + 1, memory_order_release);
    return item;
 }
};

__device__ void process_image(uchar *in, uchar *out) {
    __shared__ int histogram[256];
    __shared__ uchar map[256];

    int tid = threadIdx.x;

    if (tid < 256) {
        histogram[tid] = 0;
    }
    __syncthreads();

    for (int i = tid; i < IMG_HEIGHT * IMG_HEIGHT; i += blockDim.x)
        atomicAdd(&histogram[in[i]], 1);

    __syncthreads();

    prefix_sum(histogram, 256);

    if (tid < 256) {
        float map_value = float(histogram[tid]) / (IMG_WIDTH * IMG_HEIGHT);
        map[tid] = ((uchar)(N_COLORS * map_value)) * (256 / N_COLORS);
    }

    __syncthreads();

    for (int i = tid; i < IMG_WIDTH * IMG_HEIGHT; i += blockDim.x) {
        out[i] = map[in[i]];
    }
}


__global__ void producer_consumer_kernel(ring_buffer* cpu_to_gpu, ring_buffer* gpu_to_cpu) {
    if(threadIdx.x == 0){
        request_context req;
            do{
                req = cpu_to_gpu[blockIdx.x].pop();
                
                if(req.img_id >= 0 ){
                    process_image(req.img_in, req.img_out);
                }
                else if(context[blockIdx.x].img_id == END_RUN) {
                    return;
                }
                while(gpu_to_cpu[blockIdx.x].push(req) == failure);
            }while(1)
    }   
}

class queue_server : public image_processing_server
{
private:
    ring_buffer *cpu_to_gpu;
    ring_buffer *gpu_to_cpu;
    int n_thread_blocks;
public:
    queue_server(int threads)
    {
        char* pinned_host_buffer
        // TODO initialize host state
        // TODO launch GPU producer-consumer kernel with given number of threads
        n_thread_blocks = threads % 10; // TODO must be changed
        // Allocate pinned host buffer for two shared_memory instances
        CUDA_CHECK(hipHostMalloc(&pinned_host_buffer, 2 * sizeof(ring_buffer)));
        // Use placement new operator to construct our class on the pinned buffer
        cpu_to_gpu = new (pinned_host_buffer) ring_buffer<request_context , 4>[n_thread_blocks];
        gpu_to_cpu = new (pinned_host_buffer + sizeof(ring_buffer)) ring_buffer<request_context , 4>[n_thread_blocks];
        for(int i = 0 ; i < n_thread_blocks ; i++){
            cpu_to_gpu[i].img_id = -1;
            gpu_to_cpu[i].img_id = -1;
        }
        <<<n_thread_blocks , threads>>>process_image_kernel<<<n_thread_blocks, threads>>>(cpu_to_gpu, gpu_to_cpu);
    }

    ~queue_server() override
    {
        request_context end_context;
        end_context.img_id = END_RUN;
        std::cout << "\nKilling server" << message_from_gpu << std::endl;
        while(cpu_to_gpu[0].push(end_context) == failure);
        CUDA_CHECK( hipDeviceSynchronize() );
        delete [] cpu_to_gpu;
        delete [] gpu_to_cpu;
    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        // TODO push new task into queue if possible
        request_context req;
        req.img_id = img_id;
        req.img_in = img_in;
        req.img_out = img_out;
        for(int i = 0 ; i < n_thread_blocks ; i++){
            if(cpu_to_gpu[i].push(req) == success) return true;
        }
            return false;
    }

    bool dequeue(int *img_id) override
    {
        // TODO query (don't block) the producer-consumer queue for any responses.
        *img_id = 0; // TODO return the img_id of the request that was completed.
        for(int i = 0 ; i < n_thread_blocks ; i++){
            request_context req = gpu_to_cpu[i].pop();
            *img_id = req.img_id;
            if(req.img_id != -1) return true; 
        }
        return false;
        
    }
};

std::unique_ptr<image_processing_server> create_queues_server(int threads)
{
    return std::make_unique<queue_server>(threads);
}
