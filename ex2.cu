#include "hip/hip_runtime.h"
#include <iostream>
#include <cuda/atomic>
#include "ex2.h"

#define N_STREAMS 64
#define STREAM_AVAILABLE -1
#define Q_SLOTS 16
#define N_REGS_PER_THREAD 32
#define SHMEM_PER_BLOCK 1297

__device__ void prefix_sum(int arr[], int arr_size) {
    int tid = threadIdx.x;
    int increment;

    for (int stride = 1; stride < min(blockDim.x, arr_size); stride *= 2) {
        if (tid >= stride && tid < arr_size) {
            increment = arr[tid - stride];
        }
        __syncthreads();
        if (tid >= stride && tid < arr_size) {
            arr[tid] += increment;
        }
        __syncthreads();
    }
}

// Example single-threadblock kernel for processing a single image.
// Feel free to change it.
__global__ void process_image_kernel(uchar *in, uchar *out) {
    __shared__ int histogram[256];
    __shared__ uchar map[256];

    int tid = threadIdx.x;

    if (tid < 256) {
        histogram[tid] = 0;
    }
    __syncthreads();

    for (int i = tid; i < IMG_HEIGHT * IMG_HEIGHT; i += blockDim.x)
        atomicAdd(&histogram[in[i]], 1);

    __syncthreads();

    prefix_sum(histogram, 256);

    if (tid < 256) {
        float map_value = float(histogram[tid]) / (IMG_WIDTH * IMG_HEIGHT);
        map[tid] = ((uchar)(N_COLORS * map_value)) * (256 / N_COLORS);
    }

    __syncthreads();

    for (int i = tid; i < IMG_WIDTH * IMG_HEIGHT; i += blockDim.x) {
        out[i] = map[in[i]];
    }
}




class streams_server : public image_processing_server
{
private:
    // TODO define stream server context (memory buffers, streams, etc...)
	hipStream_t streams[N_STREAMS];

    // Feel free to change the existing memory buffer definitions.
    int img_id_in_streams[N_STREAMS];
	uchar *dimg_in;
    uchar *dimg_out;

public:
    streams_server()
    {
        // TODO initialize context (memory buffers, streams, etc...)
		
		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			img_id_in_streams[stream_idx] = STREAM_AVAILABLE;
			CUDA_CHECK(hipStreamCreate(&streams[stream_idx]));
		}
		
        CUDA_CHECK( hipMalloc(&dimg_in, N_STREAMS * IMG_WIDTH * IMG_HEIGHT) );
        CUDA_CHECK( hipMalloc(&dimg_out, N_STREAMS * IMG_WIDTH * IMG_HEIGHT) );
    }

    ~streams_server() override
    {
        // TODO free resources allocated in constructor
		
		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			CUDA_CHECK(hipStreamDestroy(streams[stream_idx]));
		}
		
		CUDA_CHECK( hipFree(dimg_in) );
        CUDA_CHECK( hipFree(dimg_out) );
    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        // TODO place memory transfers and kernel invocation in streams if possible.
		
		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			if (img_id_in_streams[stream_idx] != STREAM_AVAILABLE) {
				continue;
			}
			img_id_in_streams[stream_idx] = img_id;
			int offset = stream_idx * IMG_WIDTH * IMG_HEIGHT;
			CUDA_CHECK( hipMemcpyAsync(dimg_in + offset, img_in, IMG_WIDTH * IMG_HEIGHT, hipMemcpyHostToDevice, streams[stream_idx]));
			process_image_kernel<<<1, 1024, 0, streams[stream_idx]>>>(dimg_in + offset, dimg_out + offset);
			CUDA_CHECK( hipMemcpyAsync(img_out, dimg_out + offset, IMG_WIDTH * IMG_HEIGHT, hipMemcpyDeviceToHost, streams[stream_idx]));
//			printf("img_id: %d was enqueued to stream idx: %d\n",img_id,stream_idx);
			return true;
		}
		return false;
    }

    bool dequeue(int *img_id) override
    {
        // TODO query (don't block) streams for any completed requests.

		for (int stream_idx = 0; stream_idx < N_STREAMS; stream_idx++) {
			hipError_t status = hipStreamQuery(streams[stream_idx]);
			switch (status) {
			case hipSuccess:
				*img_id = img_id_in_streams[stream_idx]; // TODO return the img_id of the request that was completed.
				if(*img_id == STREAM_AVAILABLE) {
					continue;
				}
				img_id_in_streams[stream_idx] = STREAM_AVAILABLE;
//				printf("img_id: %d finished and was dequeued from stream idx: %d\n",*img_id,stream_idx);
				return true;
			case hipErrorNotReady:
				continue;
			default:
				CUDA_CHECK(status);
			}
		}
		return false;
    }
};

std::unique_ptr<image_processing_server> create_streams_server()
{
    return std::make_unique<streams_server>();
}

typedef struct request{
    int img_id ;
    uchar* img_in;
    uchar* img_out;
} request_context;



class ring_buffer{
private:
    static const size_t N = Q_SLOTS;
    request_context _mailbox[N];
    cuda::atomic<size_t> _head;
    cuda::atomic<size_t> _tail;
    request_context failure;
public:
    ring_buffer(){
        _head = 0;
        _tail = 0;
        failure.img_id = -1;
        for(size_t i = 0 ; i < N ; i++){
            _mailbox[i].img_id = -1;
        }
    }
    __device__ __host__ bool push(request_context data){
        size_t tail = _tail.load(cuda::memory_order_relaxed);
        if(tail - _head.load(cuda::memory_order_acquire) == N) return false;
        _mailbox[_tail % N] = data;
        _tail.store(tail + 1 , cuda::memory_order_release);
        return true;
    }
    __device__ __host__ request_context pop(){
        request_context item;
        size_t head = _head.load(cuda::memory_order_relaxed);
        if(_tail.load(cuda::memory_order_acquire) == head) return failure;
        item = _mailbox[_head % N];
        _head.store(head + 1 , cuda::memory_order_release);
        return item;
    }
};


__device__ void process_image(uchar *in, uchar *out) {
    __shared__ int histogram[256];
    __shared__ uchar map[256];

    int tid = threadIdx.x;

    if (tid < 256) {
        histogram[tid] = 0;
    }
    __syncthreads();

    for (int i = tid; i < IMG_HEIGHT * IMG_HEIGHT; i += blockDim.x)
        atomicAdd(&histogram[in[i]], 1);

    __syncthreads();

    prefix_sum(histogram, 256);

    if (tid < 256) {
        float map_value = float(histogram[tid]) / (IMG_WIDTH * IMG_HEIGHT);
        map[tid] = ((uchar)(N_COLORS * map_value)) * (256 / N_COLORS);
    }

    __syncthreads();

    for (int i = tid; i < IMG_WIDTH * IMG_HEIGHT; i += blockDim.x) {
        out[i] = map[in[i]];
    }
}


__global__ void producer_consumer_kernel(ring_buffer* cpu_to_gpu, ring_buffer* gpu_to_cpu , bool* end_run) {
    request_context req;
    __shared__ uchar* img_in;
    __shared__ uchar* img_out;
    __shared__ bool bad_request;
    do{
        if(threadIdx.x == 0){
            req = cpu_to_gpu[blockIdx.x].pop();
            bad_request = false;
            img_in = req.img_in;
            img_out = req.img_out;
            if(req.img_id == -1) bad_request = true;            
        }
        __syncthreads();
        if(bad_request) continue;   
        process_image(img_in, img_out);
        __syncthreads(); 
        if(threadIdx.x == 0){
            while(!gpu_to_cpu[blockIdx.x].push(req));
        }
    }while(!(*end_run));
    
}



int getNumOfBlocks(int threads) {
    hipDeviceProp_t prop;
    int min_limit;
    int n_used_block_regs = threads * N_REGS_PER_THREAD; 
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    int shmem_block_limit = prop.sharedMemPerMultiprocessor / SHMEM_PER_BLOCK;
    int regs_block_limit = prop.regsPerMultiprocessor / n_used_block_regs;
    int thread_block_limit = prop.maxThreadsPerMultiProcessor / threads;
    min_limit = shmem_block_limit < regs_block_limit ? shmem_block_limit : regs_block_limit;
    min_limit = min_limit < thread_block_limit ? min_limit :thread_block_limit;
    return min_limit * prop.multiProcessorCount;
}

class queue_server : public image_processing_server
{
private:
    ring_buffer *cpu_to_gpu;
    ring_buffer *gpu_to_cpu;
    char* pinned_host_buffer;
    int n_thread_blocks;
    bool* end_run;
    int last_block_idx_push;
    int last_block_idx_pop;
public:
    queue_server(int threads)
    {
        last_block_idx_push = 0;
        last_block_idx_pop = 0;
        n_thread_blocks = getNumOfBlocks(threads);
//		printf("n_thread_blocks: %d\n", n_thread_blocks);
        CUDA_CHECK(hipHostMalloc(&pinned_host_buffer, n_thread_blocks * 2 * sizeof(ring_buffer)));
        CUDA_CHECK(hipHostMalloc(&end_run, sizeof(bool)));
        *end_run = false;
        cpu_to_gpu = new (pinned_host_buffer) ring_buffer[n_thread_blocks];
        gpu_to_cpu = new (pinned_host_buffer + n_thread_blocks * sizeof(ring_buffer)) ring_buffer[n_thread_blocks]; 
        producer_consumer_kernel<<<n_thread_blocks , threads>>>(cpu_to_gpu, gpu_to_cpu, end_run);
    }

    ~queue_server() override
    {
        *end_run = true;
        CUDA_CHECK( hipDeviceSynchronize() );
        CUDA_CHECK( hipHostFree(pinned_host_buffer) );
        CUDA_CHECK( hipHostFree(end_run) );

    }

    bool enqueue(int img_id, uchar *img_in, uchar *img_out) override
    {
        request_context req;
        req.img_id = img_id;
        req.img_in = img_in;
        req.img_out = img_out;
        
        for(int i = 0 , idx = last_block_idx_push; i < n_thread_blocks ; i++ ){ 
            if(cpu_to_gpu[idx].push(req)){
                last_block_idx_push = (last_block_idx_push + 1) % n_thread_blocks;
                return true;
            }
            idx = (idx + 1) % n_thread_blocks ;
        }
        last_block_idx_push = (last_block_idx_push + 1) % n_thread_blocks;
        return false;
    }

    bool dequeue(int *img_id) override
    {
        for(int i = 0 , idx = last_block_idx_pop ; i < n_thread_blocks ; i++){         
            request_context req = gpu_to_cpu[idx].pop();
            *img_id = req.img_id;
            if(req.img_id >= 0) {
                last_block_idx_pop = (last_block_idx_pop + 1) % n_thread_blocks;
                return true; 
            }
            idx = (idx + 1) % n_thread_blocks ;
        }
        last_block_idx_pop = (last_block_idx_pop + 1) % n_thread_blocks;
        return false;
        
    }
};

std::unique_ptr<image_processing_server> create_queues_server(int threads)
{
    return std::make_unique<queue_server>(threads);
}
